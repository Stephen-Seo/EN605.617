#include "helpers.h"
#include "kernel.h"

#include <cstdlib>
#include <ctime>

void Helpers::setUpHostMemory(int **host_a, int **host_b, int **host_out,
                              unsigned int block_size,
                              unsigned int thread_size) {
    hipHostAlloc((void**)host_a, sizeof(int) * block_size * thread_size,
                  hipHostMallocDefault);
    hipHostAlloc((void**)host_b, sizeof(int) * block_size * thread_size,
                  hipHostMallocDefault);
    hipHostAlloc((void**)host_out, sizeof(int) * block_size * thread_size,
                  hipHostMallocDefault);

    srand(time(nullptr));
    for(unsigned int i = 0; i < block_size * thread_size; ++i) {
        (*host_a)[i] = i;
        (*host_b)[i] = rand() % 4;
    }
}

void Helpers::setUpDeviceMemory(int **device_a, int **device_b,
                                int **device_out,
                                unsigned int block_size,
                                unsigned int thread_size) {
    hipMalloc((void**)device_a, sizeof(int) * block_size * thread_size);
    hipMalloc((void**)device_b, sizeof(int) * block_size * thread_size);
    hipMalloc((void**)device_out, sizeof(int) * block_size * thread_size);
}

void Helpers::cleanupHostMemory(int **host_a, int **host_b, int **host_out) {
    if (host_a && *host_a) {
        hipHostFree(*host_a);
        *host_a = nullptr;
    }
    if (host_b && *host_b) {
        hipHostFree(*host_b);
        *host_b = nullptr;
    }
    if (host_out && *host_out) {
        hipHostFree(*host_out);
        *host_out = nullptr;
    }
}

void Helpers::cleanupDeviceMemory(int **device_a, int **device_b, int **device_out) {
    if (device_a && *device_a) {
        hipFree(*device_a);
        *device_a = nullptr;
    }
    if (device_b && *device_b) {
        hipFree(*device_b);
        *device_b = nullptr;
    }
    if (device_out && *device_out) {
        hipFree(*device_out);
        *device_out = nullptr;
    }
}

void Helpers::setUpStreamAndEvents(hipStream_t *stream,
                                   hipEvent_t *event_start,
                                   hipEvent_t *event_end) {
    hipStreamCreate(stream);
    hipEventCreate(event_start);
    hipEventCreate(event_end);
}

void Helpers::cleanupStreamAndEvents(hipStream_t stream,
                                     hipEvent_t event_start,
                                     hipEvent_t event_end) {
    hipStreamDestroy(stream);
    hipEventDestroy(event_start);
    hipEventDestroy(event_end);
}

void Helpers::asyncMemcpyToDevice(int *host_a, int *host_b,
                                  int *device_a, int *device_b,
                                  hipStream_t stream,
                                  hipEvent_t event_start,
                                  unsigned int block_size,
                                  unsigned int thread_size) {
    hipEventRecord(event_start);
    hipMemcpyAsync(device_a, host_a, sizeof(int) * block_size * thread_size,
                    hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(device_b, host_b, sizeof(int) * block_size * thread_size,
                    hipMemcpyHostToDevice, stream);
}

void Helpers::invokeKernel(int *device_a, int *device_b, int *device_out,
                           unsigned int block_size, unsigned int thread_size,
                           hipStream_t stream) {
    mathexpressions_events_and_streams<<<block_size, thread_size, 0, stream>>>
            (device_a, device_b, device_out);
}

void Helpers::asyncMemcpyToHost(int *host_out, int *device_out,
                                hipStream_t stream, hipEvent_t event_end,
                                unsigned int block_size,
                                unsigned int thread_size) {
    hipMemcpyAsync(host_out, device_out,
                    sizeof(int) * block_size * thread_size,
                    hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    hipEventRecord(event_end);
    hipEventSynchronize(event_end);
}

void Helpers::getEventElapsedTime(hipEvent_t event_start,
                                  hipEvent_t event_end,
                                  float *time_out) {
    hipEventElapsedTime(time_out, event_start, event_end);
}
