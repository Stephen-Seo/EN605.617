#include "arg_parse.h"
#include "pgm_rw.h"

#include <iostream>

#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <npp.h>
#include <nppdefs.h>

int main(int argc, char **argv) {
  Args args;

  if (args.ParseArgs(argc, argv)) {
    return 0;
  }

  PGMFile inputPGM;

  if (!inputPGM.LoadImage(args.input_filename)) {
    std::cout << "Failed to LoadImage" << std::endl;
    return 1;
  }

  std::vector<std::uint8_t> host_vec(inputPGM.GetSize());

  NppiRect srcROI{0, 0, (int)inputPGM.GetWidth(), (int)inputPGM.GetHeight()};

  std::uint8_t *device_in;
  std::uint8_t *device_out;
  hipMalloc(&device_in, inputPGM.GetSize());
  hipMalloc(&device_out, inputPGM.GetSize());

  hipMemcpy(device_in, inputPGM.GetImageData(), inputPGM.GetSize(),
             hipMemcpyHostToDevice);

  NppStatus status = nppiRotate_8u_C1R(
      device_in,                                              // pSrc
      {(int)inputPGM.GetWidth(), (int)inputPGM.GetHeight()},  // oSrcSize
      inputPGM.GetWidth(),                                    // nSrcStep
      srcROI,                                                 // oSrcROI
      device_out,                                             // pDst
      inputPGM.GetWidth(),                                    // nDstStep
      srcROI,                                                 // oDstROI
      180.0,                                                  // nAngle
      inputPGM.GetWidth(),                                    // nShiftX
      inputPGM.GetHeight(),                                   // nShiftY
      NPPI_INTER_NN);

  if (status != 0) {
    std::cout << "WARNING: NPP operation did not return success" << std::endl;
  }

  // hipDeviceSynchronize();

  hipMemcpy(host_vec.data(), device_out, inputPGM.GetSize(),
             hipMemcpyDeviceToHost);

  PGMFile outputPGM(host_vec.data(), host_vec.size(), inputPGM.GetWidth());
  outputPGM.SaveImage("test.pgm");

  hipFree(device_in);
  hipFree(device_out);

  return 0;
}
