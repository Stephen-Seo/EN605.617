#include "helpers.h"

#include <stdio.h>
#include <string.h>

#include "constants.h"

bool checkError(hipError_t hipError_t) {
    if (hipError_t != hipSuccess) {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(hipError_t));
        return true;
    }
    return false;
}

void allocAndSetupHostMemory(int **hostX, int **hostY, int **hostOut) {
    if (!hostX || !hostY || !hostOut) {
        return;
    }

    *hostX = (int*)malloc(sizeof(int) * TOTAL_THREADS * BLOCK_SIZE);
    *hostY = (int*)malloc(sizeof(int) * TOTAL_THREADS * BLOCK_SIZE);
    *hostOut = (int*)malloc(sizeof(int) * TOTAL_THREADS * BLOCK_SIZE);

    srand(time(NULL));
    for(unsigned int i = 0; i < TOTAL_THREADS * BLOCK_SIZE; ++i) {
        (*hostX)[i] = i;
        (*hostY)[i] = rand() % 4;
        (*hostOut)[i] = 0;
    }
}

void freeHostMemory(int **hostX, int **hostY, int **hostOut) {
    if (hostX && *hostX) {
        free(*hostX);
        *hostX = NULL;
    }
    if (hostY && *hostY) {
        free(*hostY);
        *hostY = NULL;
    }
    if (hostOut && *hostOut) {
        free(*hostOut);
        *hostOut = NULL;
    }
}

void allocDeviceMemory(int **x, int **y, int **out) {
    if (!x || !y || !out) {
        return;
    }
    hipMalloc((void**)x, sizeof(int) * TOTAL_THREADS * BLOCK_SIZE);
    checkError(hipPeekAtLastError());
    hipMalloc((void**)y, sizeof(int) * TOTAL_THREADS * BLOCK_SIZE);
    checkError(hipPeekAtLastError());
    hipMalloc((void**)out, sizeof(int) * TOTAL_THREADS * BLOCK_SIZE);
    checkError(hipPeekAtLastError());
}

void freeDeviceMemory(int **x, int **y, int **out) {
    if (x && *x) {
        hipFree(*x);
        checkError(hipPeekAtLastError());
        *x = NULL;
    }
    if (y && *y) {
        hipFree(*y);
        checkError(hipPeekAtLastError());
        *y = NULL;
    }
    if (out && *out) {
        hipFree(*out);
        checkError(hipPeekAtLastError());
        *out = NULL;
    }
}

void allocAndSetupPinnedMemory(int **x, int **y, int **out) {
    if (!x || !y || !out) {
        return;
    }

    hipHostAlloc((void**)x, sizeof(int) * TOTAL_THREADS * BLOCK_SIZE, 0);
    checkError(hipPeekAtLastError());
    hipHostAlloc((void**)y, sizeof(int) * TOTAL_THREADS * BLOCK_SIZE, 0);
    checkError(hipPeekAtLastError());
    hipHostAlloc((void**)out, sizeof(int) * TOTAL_THREADS * BLOCK_SIZE, 0);
    checkError(hipPeekAtLastError());

    srand(time(NULL));
    for(unsigned int i = 0; i < TOTAL_THREADS * BLOCK_SIZE; ++i) {
        (*x)[i] = i;
        (*y)[i] = rand() % 4;
        (*out)[i] = 0;
    }
}

void freePinnedMemory(int **x, int **y, int **out) {
    if (x && *x) {
        hipHostFree(*x);
        checkError(hipPeekAtLastError());
        *x = NULL;
    }
    if (y && *y) {
        hipHostFree(*y);
        checkError(hipPeekAtLastError());
        *y = NULL;
    }
    if (out && *out) {
        hipHostFree(*out);
        checkError(hipPeekAtLastError());
        *out = NULL;
    }
}

void hostToDeviceXY(int *hostX, int *hostY, int *deviceX, int *deviceY) {
    if (!hostX || !hostY || !deviceX || !deviceY) {
        return;
    }
    hipMemcpy(deviceX, hostX, sizeof(int) * TOTAL_THREADS * BLOCK_SIZE,
            hipMemcpyHostToDevice);
    checkError(hipPeekAtLastError());
    hipMemcpy(deviceY, hostY, sizeof(int) * TOTAL_THREADS * BLOCK_SIZE,
            hipMemcpyHostToDevice);
    checkError(hipPeekAtLastError());
}

void deviceToHostOut(int *hostOut, int *deviceOut) {
    if (!hostOut || !deviceOut) {
        return;
    }
    hipMemcpy(hostOut, deviceOut, sizeof(int) * TOTAL_THREADS * BLOCK_SIZE,
            hipMemcpyDeviceToHost);
}

void printHostOut(int *hostOut) {
    for(unsigned int j = 0; j <= TOTAL_THREADS * BLOCK_SIZE / 4; ++j) {
        if (j * 4 < TOTAL_THREADS * BLOCK_SIZE) {
            printf("%4u: %4d\t", j * 4, hostOut[j * 4]);
            if (1 + j * 4 < TOTAL_THREADS * BLOCK_SIZE) {
                printf("%4u: %4d\t", 1 + j * 4, hostOut[1 + j * 4]);
                if (2 + j * 4 < TOTAL_THREADS * BLOCK_SIZE) {
                    printf("%4u: %4d\t", 2 + j * 4, hostOut[2 + j * 4]);
                    if (3 + j * 4 < TOTAL_THREADS * BLOCK_SIZE) {
                        printf("%4u: %4d\n", 3 + j * 4, hostOut[3 + j * 4]);
                    } else {
                        printf("\n");
                        break;
                    }
                } else {
                    printf("\n");
                    break;
                }
            } else {
                printf("\n");
                break;
            }
        } else {
            printf("\n");
            break;
        }
    }
}

void cipher_allocAndSetupHostMemory(char **host) {
    if (!host) {
        return;
    }

    *host = (char*)malloc(sizeof(char) * TOTAL_THREADS * BLOCK_SIZE);

    unsigned int i = 0;
    for (; i < TOTAL_THREADS * BLOCK_SIZE; i += CYPHER_PHRASE_SIZE) {
        memcpy(*host + i, CYPHER_PHRASE, sizeof(char) * 26);
    }
    if (i > TOTAL_THREADS * BLOCK_SIZE) {
        i -= CYPHER_PHRASE_SIZE;
        memcpy(*host + i, CYPHER_PHRASE, sizeof(char) * (TOTAL_THREADS * BLOCK_SIZE - i));
    }
}

void cipher_freeHostMemory(char **host) {
    if (host && *host) {
        free(*host);
        *host = NULL;
    }
}

void cipher_allocDeviceMemory(char **device) {
    if(!device) {
        return;
    }
    hipMalloc((void**)device, sizeof(char) * TOTAL_THREADS * BLOCK_SIZE);
    checkError(hipPeekAtLastError());
}

void cipher_freeDeviceMemory(char **device) {
    if(device && *device) {
        hipFree(*device);
        checkError(hipPeekAtLastError());
        *device = NULL;
    }
}

void cipher_hostToDevice(char *host, char *device) {
    if(!host || !device) {
        return;
    }
    hipMemcpy(device, host, sizeof(char) * TOTAL_THREADS * BLOCK_SIZE,
            hipMemcpyHostToDevice);
    checkError(hipPeekAtLastError());
}

void cipher_deviceToHost(char *host, char *device) {
    if(!host || !device) {
        return;
    }
    hipMemcpy(host, device, sizeof(char) * TOTAL_THREADS * BLOCK_SIZE,
            hipMemcpyDeviceToHost);
    checkError(hipPeekAtLastError());
}

void cipher_printChars(char *host) {
    unsigned int j = 0;
    for (; j * 64 < TOTAL_THREADS * BLOCK_SIZE; ++j) {
        for (unsigned int i = 0; i < 64; ++i) {
            printf("%c", host[i + j * 64]);
        }
        printf("\n");
    }
    if (j * 64 < TOTAL_THREADS * BLOCK_SIZE) {
        for (unsigned int i = j * 64; i < TOTAL_THREADS * BLOCK_SIZE; ++i) {
            printf("%c", host[i]);
        }
        printf("\n");
    }
}

// vim: cindent: ts=4: sw=4: et
