#include "helpers.h"

#include <stdio.h>

#include "constants.h"

bool checkError(hipError_t hipError_t) {
    if (hipError_t != hipSuccess) {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(hipError_t));
        return true;
    }
    return false;
}

void allocAndSetupHostMemory(int **hostX, int **hostY, int **hostOut) {
    if (!hostX || !hostY || !hostOut) {
        return;
    }

    *hostX = (int*)malloc(sizeof(int) * totalThreads);
    *hostY = (int*)malloc(sizeof(int) * totalThreads);
    *hostOut = (int*)malloc(sizeof(int) * totalThreads);

    srand(time(NULL));
    for(unsigned int i = 0; i < totalThreads; ++i) {
        (*hostX)[i] = i;
        (*hostY)[i] = rand() % 4;
        (*hostOut)[i] = 0;
    }
}

void freeHostMemory(int *hostX, int *hostY, int *hostOut) {
    free(hostX);
    free(hostY);
    free(hostOut);
}

void allocDeviceMemory(int **x, int **y, int **out) {
    hipMalloc((void**)x, sizeof(int) * totalThreads);
    checkError(hipPeekAtLastError());
    hipMalloc((void**)y, sizeof(int) * totalThreads);
    checkError(hipPeekAtLastError());
    hipMalloc((void**)out, sizeof(int) * totalThreads);
    checkError(hipPeekAtLastError());
}

void freeDeviceMemory(int *x, int *y, int *out) {
    hipFree(x);
    checkError(hipPeekAtLastError());
    hipFree(y);
    checkError(hipPeekAtLastError());
    hipFree(out);
    checkError(hipPeekAtLastError());
}

void allocAndSetupPinnedMemory(int **x, int **y, int **out) {
    hipHostAlloc((void**)x, sizeof(int) * totalThreads, 0);
    checkError(hipPeekAtLastError());
    hipHostAlloc((void**)y, sizeof(int) * totalThreads, 0);
    checkError(hipPeekAtLastError());
    hipHostAlloc((void**)out, sizeof(int) * totalThreads, 0);
    checkError(hipPeekAtLastError());

    srand(time(NULL));
    for(unsigned int i = 0; i < totalThreads; ++i) {
        (*x)[i] = i;
        (*y)[i] = rand() % 4;
        (*out)[i] = 0;
    }
}

void freePinnedMemory(int *x, int *y, int *out) {
    hipHostFree(x);
    checkError(hipPeekAtLastError());
    hipHostFree(y);
    checkError(hipPeekAtLastError());
    hipHostFree(out);
    checkError(hipPeekAtLastError());
}

void hostToDeviceXY(int *hostX, int *hostY, int *deviceX, int *deviceY) {
    hipMemcpy(deviceX, hostX, sizeof(int) * totalThreads,
            hipMemcpyHostToDevice);
    checkError(hipPeekAtLastError());
    hipMemcpy(deviceY, hostY, sizeof(int) * totalThreads,
            hipMemcpyHostToDevice);
    checkError(hipPeekAtLastError());
}

void deviceToHostOut(int *hostOut, int *deviceOut) {
    hipMemcpy(hostOut, deviceOut, sizeof(int) * totalThreads,
            hipMemcpyDeviceToHost);
}

void printHostOut(int *hostOut) {
    for(unsigned int j = 0; j <= totalThreads / 4; ++j) {
        if (j * 4 < totalThreads) {
            printf("%4u: %4d\t", j * 4, hostOut[j * 4]);
            if (1 + j * 4 < totalThreads) {
                printf("%4u: %4d\t", 1 + j * 4, hostOut[1 + j * 4]);
                if (2 + j * 4 < totalThreads) {
                    printf("%4u: %4d\t", 2 + j * 4, hostOut[2 + j * 4]);
                    if (3 + j * 4 < totalThreads) {
                        printf("%4u: %4d\n", 3 + j * 4, hostOut[3 + j * 4]);
                    } else {
                        printf("\n");
                        break;
                    }
                } else {
                    printf("\n");
                    break;
                }
            } else {
                printf("\n");
                break;
            }
        } else {
            printf("\n");
            break;
        }
    }
}
