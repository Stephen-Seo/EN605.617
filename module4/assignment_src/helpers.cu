#include "helpers.h"

#include <stdio.h>
#include <string.h>

#include "constants.h"

bool checkError(hipError_t hipError_t) {
    if (hipError_t != hipSuccess) {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(hipError_t));
        return true;
    }
    return false;
}

void allocAndSetupHostMemory(int **hostX, int **hostY, int **hostOut) {
    if (!hostX || !hostY || !hostOut) {
        return;
    }

    *hostX = (int*)malloc(sizeof(int) * totalThreads);
    *hostY = (int*)malloc(sizeof(int) * totalThreads);
    *hostOut = (int*)malloc(sizeof(int) * totalThreads);

    srand(time(NULL));
    for(unsigned int i = 0; i < totalThreads; ++i) {
        (*hostX)[i] = i;
        (*hostY)[i] = rand() % 4;
        (*hostOut)[i] = 0;
    }
}

void freeHostMemory(int **hostX, int **hostY, int **hostOut) {
    if (hostX && *hostX) {
        free(*hostX);
        *hostX = NULL;
    }
    if (hostY && *hostY) {
        free(*hostY);
        *hostY = NULL;
    }
    if (hostOut && *hostOut) {
        free(*hostOut);
        *hostOut = NULL;
    }
}

void allocDeviceMemory(int **x, int **y, int **out) {
    if (!x || !y || !out) {
        return;
    }
    hipMalloc((void**)x, sizeof(int) * totalThreads);
    checkError(hipPeekAtLastError());
    hipMalloc((void**)y, sizeof(int) * totalThreads);
    checkError(hipPeekAtLastError());
    hipMalloc((void**)out, sizeof(int) * totalThreads);
    checkError(hipPeekAtLastError());
}

void freeDeviceMemory(int **x, int **y, int **out) {
    if (x && *x) {
        hipFree(*x);
        checkError(hipPeekAtLastError());
        *x = NULL;
    }
    if (y && *y) {
        hipFree(*y);
        checkError(hipPeekAtLastError());
        *y = NULL;
    }
    if (out && *out) {
        hipFree(*out);
        checkError(hipPeekAtLastError());
        *out = NULL;
    }
}

void allocAndSetupPinnedMemory(int **x, int **y, int **out) {
    if (!x || !y || !out) {
        return;
    }

    hipHostAlloc((void**)x, sizeof(int) * totalThreads, 0);
    checkError(hipPeekAtLastError());
    hipHostAlloc((void**)y, sizeof(int) * totalThreads, 0);
    checkError(hipPeekAtLastError());
    hipHostAlloc((void**)out, sizeof(int) * totalThreads, 0);
    checkError(hipPeekAtLastError());

    srand(time(NULL));
    for(unsigned int i = 0; i < totalThreads; ++i) {
        (*x)[i] = i;
        (*y)[i] = rand() % 4;
        (*out)[i] = 0;
    }
}

void freePinnedMemory(int **x, int **y, int **out) {
    if (x && *x) {
        hipHostFree(*x);
        checkError(hipPeekAtLastError());
        *x = NULL;
    }
    if (y && *y) {
        hipHostFree(*y);
        checkError(hipPeekAtLastError());
        *y = NULL;
    }
    if (out && *out) {
        hipHostFree(*out);
        checkError(hipPeekAtLastError());
        *out = NULL;
    }
}

void hostToDeviceXY(int *hostX, int *hostY, int *deviceX, int *deviceY) {
    if (!hostX || !hostY || !deviceX || !deviceY) {
        return;
    }
    hipMemcpy(deviceX, hostX, sizeof(int) * totalThreads,
            hipMemcpyHostToDevice);
    checkError(hipPeekAtLastError());
    hipMemcpy(deviceY, hostY, sizeof(int) * totalThreads,
            hipMemcpyHostToDevice);
    checkError(hipPeekAtLastError());
}

void deviceToHostOut(int *hostOut, int *deviceOut) {
    if (!hostOut || !deviceOut) {
        return;
    }
    hipMemcpy(hostOut, deviceOut, sizeof(int) * totalThreads,
            hipMemcpyDeviceToHost);
}

void printHostOut(int *hostOut) {
    for(unsigned int j = 0; j <= totalThreads / 4; ++j) {
        if (j * 4 < totalThreads) {
            printf("%4u: %4d\t", j * 4, hostOut[j * 4]);
            if (1 + j * 4 < totalThreads) {
                printf("%4u: %4d\t", 1 + j * 4, hostOut[1 + j * 4]);
                if (2 + j * 4 < totalThreads) {
                    printf("%4u: %4d\t", 2 + j * 4, hostOut[2 + j * 4]);
                    if (3 + j * 4 < totalThreads) {
                        printf("%4u: %4d\n", 3 + j * 4, hostOut[3 + j * 4]);
                    } else {
                        printf("\n");
                        break;
                    }
                } else {
                    printf("\n");
                    break;
                }
            } else {
                printf("\n");
                break;
            }
        } else {
            printf("\n");
            break;
        }
    }
}

void cypher_allocAndSetupHostMemory(char **host) {
    if (!host) {
        return;
    }

    *host = (char*)malloc(sizeof(char) * totalThreads);

    unsigned int i = 0;
    for (; i < totalThreads; i += cypherPhraseSize) {
        memcpy(*host + i, cypherPhrase, sizeof(char) * 26);
    }
    if (i > totalThreads) {
        i -= cypherPhraseSize;
        memcpy(*host + i, cypherPhrase, sizeof(char) * (totalThreads - i));
    }
}

void cypher_freeHostMemory(char **host) {
    if (host && *host) {
        free(*host);
        *host = NULL;
    }
}

void cypher_allocDeviceMemory(char **device) {
    if(!device) {
        return;
    }
    hipMalloc((void**)device, sizeof(char) * totalThreads);
    checkError(hipPeekAtLastError());
}

void cypher_freeDeviceMemory(char **device) {
    if(device && *device) {
        hipFree(*device);
        checkError(hipPeekAtLastError());
        *device = NULL;
    }
}

void cypher_hostToDevice(char *host, char *device) {
    if(!host || !device) {
        return;
    }
    hipMemcpy(device, host, sizeof(char) * totalThreads,
            hipMemcpyHostToDevice);
    checkError(hipPeekAtLastError());
}

void cypher_deviceToHost(char *host, char *device) {
    if(!host || !device) {
        return;
    }
    hipMemcpy(host, device, sizeof(char) * totalThreads,
            hipMemcpyDeviceToHost);
    checkError(hipPeekAtLastError());
}

void cypher_printChars(char *host) {
    unsigned int j = 0;
    for (; j * 64 < totalThreads; ++j) {
        for (unsigned int i = 0; i < 64; ++i) {
            printf("%c", host[i + j * 64]);
        }
        printf("\n");
    }
    if (j * 64 < totalThreads) {
        for (unsigned int i = j * 64; i < totalThreads; ++i) {
            printf("%c", host[i]);
        }
        printf("\n");
    }
}

// vim: cindent: ts=4: sw=4: et
