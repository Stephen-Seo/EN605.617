//Based on the work of Andrew Krepps

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <chrono>

__global__
void add(int *x, int *y, int *out) {
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    out[thread_idx] = x[thread_idx] + y[thread_idx];
}

__global__
void subtract(int *x, int *y, int *out) {
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    out[thread_idx] = x[thread_idx] - y[thread_idx];
}

__global__
void mult(int *x, int *y, int *out) {
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    out[thread_idx] = x[thread_idx] * y[thread_idx];
}

__global__
void mod(int *x, int *y, int *out) {
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    out[thread_idx] = x[thread_idx] % y[thread_idx];
}

//__global__
//void add_offset(int *x, int *y, int *out) {
//    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
//    out[thread_idx * 4] = x[thread_idx * 4] + y[thread_idx * 4];
//}
//
//__global__
//void subtract_offset(int *x, int *y, int *out) {
//    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
//    out[thread_idx * 4 + 1] = x[thread_idx * 4 + 1] - y[thread_idx * 4 + 1];
//}
//
//__global__
//void mult_offset(int *x, int *y, int *out) {
//    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
//    out[thread_idx * 4 + 2] = x[thread_idx * 4 + 2] * y[thread_idx * 4 + 2];
//}
//
//__global__
//void mod_offset(int *x, int *y, int *out) {
//    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
//    out[thread_idx * 4 + 3] = x[thread_idx * 4 + 3] % y[thread_idx + 4 + 3];
//}

__global__
void branching_mathexpressions(int *x, int *y, int *out) {
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    const unsigned int subidx = thread_idx % 4;
    if (subidx == 0) {
        out[thread_idx] = x[thread_idx] + y[thread_idx];
    } else if (subidx == 1) {
        out[thread_idx] = x[thread_idx] - y[thread_idx];
    } else if (subidx == 2) {
        out[thread_idx] = x[thread_idx] * y[thread_idx];
    } else {
        out[thread_idx] = x[thread_idx] % y[thread_idx];
    }
}

__global__
void non_branching_mathexpressions(int *x, int *y, int *out) {
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    const unsigned int sidx = thread_idx % 4;
    // i=0  (3 ^ i) & 1 >> i = 1
    // i=1  (3 ^ i) & 1 >> i = 0
    // i=2  (3 ^ i) & 1 >> i = 0
    // i=3  (3 ^ i) & 1 >> i = 0
    //
    // i=0  ((3 >> i) & 1) * i = 0
    // i=1  ((3 >> i) & 1) * i = 1
    // i=2  ((3 >> i) & 1) * i = 0
    // i=3  ((3 >> i) & 1) * i = 0
    //
    // i=0  (3 ^ (3-i)) & 1 >> (3-i) = 0
    // i=1  (3 ^ (3-i)) & 1 >> (3-i) = 0
    // i=2  (3 ^ (3-i)) & 1 >> (3-i) = 0
    // i=3  (3 ^ (3-i)) & 1 >> (3-i) = 1
    //
    // i=0  ((3 >> (3-i)) & 1) * (3-i) = 0
    // i=1  ((3 >> (3-i)) & 1) * (3-i) = 0
    // i=2  ((3 >> (3-i)) & 1) * (3-i) = 1
    // i=3  ((3 >> (3-i)) & 1) * (3-i) = 0
    out[thread_idx] =
        (((3 ^ sidx) & 1) >> sidx)         * (x[thread_idx] + y[thread_idx])
      + (((3 >> sidx) & 1) * sidx)         * (x[thread_idx] - y[thread_idx])
      + (((3 >> (3-sidx)) & 1) * (3-sidx)) * (x[thread_idx] * y[thread_idx])
      + (((3 ^ (3-sidx)) & 1) >> (3-sidx)) * (x[thread_idx] % y[thread_idx]);
}

//__global__
//void branching_mulmod(int *x, int *y, int *out) {
//    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
//    if (thread_idx % 2 == 0) {
//        out[thread_idx] = x[thread_idx] * y[thread_idx];
//    } else {
//        out[thread_idx] = x[thread_idx] % y[thread_idx];
//    }
//}
//
//__global__
//void non_branching_mulmod(int *x, int *y, int *out) {
//    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
//    const unsigned int mod_result = thread_idx % 2;
//    out[thread_idx] = x[thread_idx] % y[thread_idx] * mod_result
//        + x[thread_idx] * y[thread_idx] * (1 - mod_result);
//}

enum MathFnToUse {
    MFN_ADD,
    MFN_SUB,
    MFN_MUL,
    MFN_MOD,
    MFN_BRANCHING,
    MFN_NON_BRANCHING
};

bool checkError(hipError_t hipError_t) {
    if (hipError_t != hipSuccess) {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(hipError_t));
        return true;
    }
    return false;
}

int main(int argc, char** argv) {
	// read command line arguments
	int totalThreads = 512;
	int blockSize = 256;
    MathFnToUse fn = MFN_ADD;
	
	if (argc >= 2) {
		totalThreads = atoi(argv[1]);
        fprintf(stderr, "Got input %3u for total threads\n", totalThreads);
	}
	if (argc >= 3) {
		blockSize = atoi(argv[2]);
        fprintf(stderr, "Got input %3u for blockSize\n", blockSize);
	}
    if (argc >= 4) {
        if (argv[3][0] == 'a') {
            fn = MFN_ADD;
            fprintf(stderr, "Using \"add\" fn\n");
        } else if (argv[3][0] == 's') {
            fn = MFN_SUB;
            fprintf(stderr, "Using \"subtract\" fn\n");
        } else if (argv[3][0] == 'm') {
            fn = MFN_MUL;
            fprintf(stderr, "Using \"multiply\" fn\n");
        } else if (argv[3][0] == 'o') {
            fn = MFN_MOD;
            fprintf(stderr, "Using \"modulus\" fn\n");
        } else if (argv[3][0] == 'b') {
            fn = MFN_BRANCHING;
            fprintf(stderr, "Using \"branching\" fn\n");
        } else if (argv[3][0] == 'n') {
            fn = MFN_NON_BRANCHING;
            fprintf(stderr, "Using \"non-branching\" fn\n");
        } else {
            fprintf(stderr,
                    "Invalid third argument, using \"add\" fn by default\n");
        }
    }

	int numBlocks = totalThreads/blockSize;

	// validate command line arguments
    if (totalThreads % blockSize != 0) {
		++numBlocks;
		totalThreads = numBlocks*blockSize;
		
		fprintf(stderr, "Warning: Total thread count is not evenly divisible by"
               " the block size\n");
		fprintf(stderr,
                "The total number of threads will be rounded up to %d\n",
                totalThreads);
	}
    fprintf(stderr, "totalThreads == %3u, numBlocks == %3u, blockSize == %3u\n",
        totalThreads, numBlocks, blockSize);

    int *hostX =
        (int*)malloc(sizeof(int) * totalThreads);
    int *hostY =
        (int*)malloc(sizeof(int) * totalThreads);
    int *hostOut =
        (int*)malloc(sizeof(int) * totalThreads);

    fprintf(stderr, "Setting host values...\n");
    srand(time(NULL));
    for(unsigned int i = 0; i < totalThreads; ++i) {
        hostX[i] = i;
        hostY[i] = rand() % 4;
        hostOut[i] = 0;
    }

    int *x;
    int *y;
    int *out;

    fprintf(stderr, "cudaMalloc...\n");
    hipMalloc((void**)&x, totalThreads * sizeof(int));
    checkError(hipPeekAtLastError());
    hipMalloc((void**)&y, totalThreads * sizeof(int));
    checkError(hipPeekAtLastError());
    hipMalloc((void**)&out, totalThreads * sizeof(int));
    checkError(hipPeekAtLastError());

    fprintf(stderr, "cudaMemcpy...\n");
    hipMemcpy(x, hostX, totalThreads * sizeof(int),
            hipMemcpyHostToDevice);
    checkError(hipPeekAtLastError());
    hipMemcpy(y, hostY, totalThreads * sizeof(int),
            hipMemcpyHostToDevice);
    checkError(hipPeekAtLastError());
    hipMemcpy(out, hostOut, totalThreads * sizeof(int),
            hipMemcpyHostToDevice);
    checkError(hipPeekAtLastError());

    switch (fn) {
    case MFN_ADD: {
        fprintf(stderr, "Executing \"add\"...\n");
        auto start_clock = std::chrono::high_resolution_clock::now();
        add<<<numBlocks, totalThreads>>>(x, y, out);
        hipError_t hipError_t = hipDeviceSynchronize();
        auto end_clock = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(
                    end_clock - start_clock);
        fprintf(stderr, "Duration of \"add\" nanos: %lld\n", duration.count());
        checkError(hipError_t);
      } break;
    case MFN_SUB: {
        fprintf(stderr, "Executing \"sub\"...\n");
        auto start_clock = std::chrono::high_resolution_clock::now();
        subtract<<<numBlocks, totalThreads>>>(x, y, out);
        hipError_t hipError_t = hipDeviceSynchronize();
        auto end_clock = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(
                    end_clock - start_clock);
        fprintf(stderr, "Duration of \"sub\" nanos: %lld\n", duration.count());
        checkError(hipError_t);
      } break;
    case MFN_MUL: {
        fprintf(stderr, "Executing \"mul\"...\n");
        auto start_clock = std::chrono::high_resolution_clock::now();
        mult<<<numBlocks, totalThreads>>>(x, y, out);
        hipError_t hipError_t = hipDeviceSynchronize();
        auto end_clock = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(
                    end_clock - start_clock);
        fprintf(stderr, "Duration of \"mul\" nanos: %lld\n", duration.count());
        checkError(hipError_t);
      } break;
    case MFN_MOD: {
        fprintf(stderr, "Executing \"mod\"...\n");
        auto start_clock = std::chrono::high_resolution_clock::now();
        mod<<<numBlocks, totalThreads>>>(x, y, out);
        hipError_t hipError_t = hipDeviceSynchronize();
        auto end_clock = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(
                    end_clock - start_clock);
        fprintf(stderr, "Duration of \"mod\" nanos: %lld\n", duration.count());
        checkError(hipError_t);
      } break;
    case MFN_BRANCHING: {
        fprintf(stderr, "Executing \"branching\" (%u blocks, %u threads)...\n",
                numBlocks, totalThreads);
        unsigned long long count = 0;
        for (unsigned int i = 0; i < 25; ++i) {
            auto start_clock = std::chrono::high_resolution_clock::now();
            branching_mathexpressions<<<numBlocks, totalThreads>>>(x, y, out);
            hipError_t hipError_t = hipDeviceSynchronize();
            auto end_clock = std::chrono::high_resolution_clock::now();
            auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>
                    (end_clock - start_clock);
            if (i > 4) {
                fprintf(stderr, "Duration of \"branching\" nanos: %lld\n",
                        duration.count());
                count += duration.count();
            }
            checkError(hipError_t);
        } // for
        fprintf(stderr, "Average of duration nanos == %llu\n", count / 20);
        } break;
    case MFN_NON_BRANCHING: {
        fprintf(stderr,
                "Executing \"non-branching\" (%u blocks, %u threads)...\n",
                numBlocks, totalThreads);
        unsigned long long count = 0;
        for (unsigned int i = 0; i < 25; ++i) {
            auto start_clock = std::chrono::high_resolution_clock::now();
            non_branching_mathexpressions<<<numBlocks, totalThreads>>>(x,
                                                                       y,
                                                                       out);
            hipError_t hipError_t = hipDeviceSynchronize();
            auto end_clock = std::chrono::high_resolution_clock::now();
            auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>
                    (end_clock - start_clock);
            if (i > 4) {
                fprintf(stderr, "Duration of \"non-branching\" nanos: %lld\n",
                        duration.count());
                count += duration.count();
            }
            checkError(hipError_t);
        } // for
        fprintf(stderr, "Average of duration nanos == %llu\n", count / 20);
        } break;
    default:
        fprintf(stderr, "ERROR: Invalid state\n");
        hipFree(x);
        hipFree(y);
        hipFree(out);
        free(hostX);
        free(hostY);
        free(hostOut);
        return 1;
    }

    checkError(hipPeekAtLastError());

    fprintf(stderr, "Copying result to host...\n");
    hipMemcpy(hostOut, out, totalThreads * sizeof(int),
            hipMemcpyDeviceToHost);

    fprintf(stderr, "Freeing device memory...\n");
    hipFree(x);
    hipFree(y);
    hipFree(out);

    // print results
    for(unsigned int j = 0; j <= totalThreads / 4; ++j) {
        if (j * 4 < totalThreads) {
            printf("%4u: %4d\t", j * 4, hostOut[j * 4]);
            if (1 + j * 4 < totalThreads) {
                printf("%4u: %4d\t", 1 + j * 4, hostOut[1 + j * 4]);
                if (2 + j * 4 < totalThreads) {
                    printf("%4u: %4d\t", 2 + j * 4, hostOut[2 + j * 4]);
                    if (3 + j * 4 < totalThreads) {
                        printf("%4u: %4d\n", 3 + j * 4, hostOut[3 + j * 4]);
                    } else {
                        printf("\n");
                        break;
                    }
                } else {
                    printf("\n");
                    break;
                }
            } else {
                printf("\n");
                break;
            }
        } else {
            printf("\n");
            break;
        }
    }

    fprintf(stderr, "Freeing host memory...\n");
    free(hostX);
    free(hostY);
    free(hostOut);

    return EXIT_SUCCESS;
}

// vim: cindent: ts=4: sw=4: et
