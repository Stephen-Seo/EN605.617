//Based on the work of Andrew Krepps

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__
void add(int *x, int *y, int *out) {
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    out[thread_idx] = x[thread_idx] + y[thread_idx];
}

__global__
void subtract(int *x, int *y, int *out) {
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    out[thread_idx] = x[thread_idx] - y[thread_idx];
}

__global__
void mult(int *x, int *y, int *out) {
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    out[thread_idx] = x[thread_idx] * y[thread_idx];
}

__global__
void mod(int *x, int *y, int *out) {
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    out[thread_idx] = x[thread_idx] % y[thread_idx];
}

enum MathFnToUse {
    MFN_ADD,
    MFN_SUB,
    MFN_MUL,
    MFN_MOD
};

int main(int argc, char** argv) {
	// read command line arguments
	int totalThreads = 512;
	int blockSize = 256;
    MathFnToUse fn = MFN_ADD;
	
	if (argc >= 2) {
		totalThreads = atoi(argv[1]);
        printf("Got input %3u for total threads\n", totalThreads);
	}
	if (argc >= 3) {
		blockSize = atoi(argv[2]);
        printf("Got input %3u for blockSize\n", blockSize);
	}
    if (argc >= 4) {
        if (argv[3][0] == 'a') {
            fn = MFN_ADD;
            printf("Using \"add\" fn\n");
        } else if (argv[3][0] == 's') {
            fn = MFN_SUB;
            printf("Using \"subtract\" fn\n");
        } else if (argv[3][0] == 'm') {
            fn = MFN_MUL;
            printf("Using \"multiply\" fn\n");
        } else if (argv[3][0] == 'o') {
            fn = MFN_MOD;
            printf("Using \"modulus\" fn\n");
        } else {
            printf("Invalid third argument, using \"add\" fn by default\n");
        }
    }

	int numBlocks = totalThreads/blockSize;

	// validate command line arguments
	if (totalThreads % blockSize != 0) {
		++numBlocks;
		totalThreads = numBlocks*blockSize;
		
		printf("Warning: Total thread count is not evenly divisible by the "
               "block size\n");
		printf("The total number of threads will be rounded up to %d\n",
                totalThreads);
	}
    printf("totalThreads == %3u, numBlocks == %3u, blockSize == %3u\n",
        totalThreads, numBlocks, blockSize);

    int *hostX =
        (int*)malloc(sizeof(int) * totalThreads);
    int *hostY =
        (int*)malloc(sizeof(int) * totalThreads);
    int *hostOut =
        (int*)malloc(sizeof(int) * totalThreads);

    printf("Setting host values...\n");
    srand(time(NULL));
    for(unsigned int i = 0; i < totalThreads; ++i) {
        hostX[i] = i;
        hostY[i] = rand() % 4;
        // hostOut will be overwritten by data output from CUDA
        //hostOut[i] = 0;
    }

    int *x;
    int *y;
    int *out;

    printf("cudaMalloc...\n");
    hipMalloc((void**)&x, totalThreads * sizeof(int));
    hipMalloc((void**)&y, totalThreads * sizeof(int));
    hipMalloc((void**)&out, totalThreads * sizeof(int));

    printf("cudaMemcpy...\n");
    hipMemcpy(x, hostX, totalThreads * sizeof(int),
            hipMemcpyHostToDevice);
    hipMemcpy(y, hostY, totalThreads * sizeof(int),
            hipMemcpyHostToDevice);

    switch (fn) {
    case MFN_ADD:
        printf("Executing \"add\"...\n");
        add<<<numBlocks, totalThreads>>>(x, y, out);
        break;
    case MFN_SUB:
        printf("Executing \"sub\"...\n");
        subtract<<<numBlocks, totalThreads>>>(x, y, out);
        break;
    case MFN_MUL:
        printf("Executing \"mul\"...\n");
        mult<<<numBlocks, totalThreads>>>(x, y, out);
        break;
    case MFN_MOD:
        printf("Executing \"mod\"...\n");
        mod<<<numBlocks, totalThreads>>>(x, y, out);
        break;
    default:
        printf("ERROR: Invalid state\n");
        hipFree(x);
        hipFree(y);
        hipFree(out);
        free(hostX);
        free(hostY);
        free(hostOut);
        return 1;
    }

    printf("Copying result to host...\n");
    hipMemcpy(hostOut, out, totalThreads * sizeof(int),
            hipMemcpyDeviceToHost);

    printf("Freeing device memory...\n");
    hipFree(x);
    hipFree(y);
    hipFree(out);

    for(unsigned int j = 0; j <= totalThreads / 4; ++j) {
        if (j * 4 < totalThreads) {
            printf("%4u: %4d\t", j * 4, hostOut[j * 4]);
            if (1 + j * 4 < totalThreads) {
                printf("%4u: %4d\t", 1 + j * 4, hostOut[1 + j * 4]);
                if (2 + j * 4 < totalThreads) {
                    printf("%4u: %4d\t", 2 + j * 4, hostOut[2 + j * 4]);
                    if (3 + j * 4 < totalThreads) {
                        printf("%4u: %4d\n", 3 + j * 4, hostOut[3 + j * 4]);
                    } else {
                        printf("\n");
                        break;
                    }
                } else {
                    printf("\n");
                    break;
                }
            } else {
                printf("\n");
                break;
            }
        } else {
            printf("\n");
            break;
        }
    }

    printf("Freeing host memory...\n");
    free(hostX);
    free(hostY);
    free(hostOut);

    return EXIT_SUCCESS;
}

// vim: cindent: ts=4: sw=4: et
