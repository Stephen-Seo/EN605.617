#include "hip/hip_runtime.h"
#include "cuda_rand_kernel.h"

#include "constants.h"

__global__
void GenerateRandom(hiprandState_t *state, unsigned int seed, unsigned int *out) {
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    hiprand_init(seed,
                thread_idx,
                0,
                &state[thread_idx]);

    out[thread_idx] = hiprand(&state[thread_idx]) % kMaxRand;
}
