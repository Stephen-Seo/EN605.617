#include "helpers.h"

void helpers::InitRandStates(hiprandState_t **states,
                               unsigned int num_blocks,
                               unsigned int num_threads) {
    hipMalloc(states, num_blocks * num_threads * sizeof(hiprandState_t));
}

void helpers::FreeRandStates(hiprandState_t **states) {
    if(states && *states) {
        hipFree(*states);
        *states = nullptr;
    }
}
